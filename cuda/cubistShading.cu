#include "hip/hip_runtime.h"
//
// Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions
// are met:
//  * Redistributions of source code must retain the above copyright
//    notice, this list of conditions and the following disclaimer.
//  * Redistributions in binary form must reproduce the above copyright
//    notice, this list of conditions and the following disclaimer in the
//    documentation and/or other materials provided with the distribution.
//  * Neither the name of NVIDIA CORPORATION nor the names of its
//    contributors may be used to endorse or promote products derived
//    from this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
// EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
// PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
// CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
// EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
// PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
// PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
// OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
// (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
//
#include <optix.h>

#include "LocalGeometry.h"
#include "helpers.h"
#include "random.h"
#include "../cubistutil/vec_math.h"

#include "cubistShading.h"

//------------------------------------------------------------------------------
//
//
//
//------------------------------------------------------------------------------

extern "C" __global__ void __raygen__pinhole()
{
    const uint3  launch_idx     = optixGetLaunchIndex();
    const uint3  launch_dims    = optixGetLaunchDimensions();
    const float3 eye            = cubist::params.eye;
    const float3 U              = cubist::params.U;
    const float3 V              = cubist::params.V;
    const float3 W              = cubist::params.W;
    const int    subframe_index = cubist::params.subframe_index;

    //
    // Generate camera ray
    //
    unsigned int seed = tea<4>( launch_idx.y * launch_dims.x + launch_idx.x, subframe_index );

    // The center of each pixel is at fraction (0.5,0.5)
    const float2 subpixel_jitter =
        subframe_index == 0 ? make_float2( 0.5f, 0.5f ) : make_float2( rnd( seed ), rnd( seed ) );

    const float2 d =
        2.0f
            * make_float2( ( static_cast<float>( launch_idx.x ) + subpixel_jitter.x ) / static_cast<float>( launch_dims.x ),
                           ( static_cast<float>( launch_idx.y ) + subpixel_jitter.y ) / static_cast<float>( launch_dims.y ) )
        - 1.0f;
    const float3 ray_direction = normalize( d.x * U + d.y * V + W );
    const float3 ray_origin    = eye;

    //
    // Trace camera ray
    //
    cubist::PayloadRadiance payload;
    payload.result          = make_float3( 0.0f );
    payload.importance      = 1.0f;
    payload.depth           = 0.0f;

    // trace first cubist pass
    // the first pass will determine whether to trace the second pass or not.
    bool isSecondCubist = traceFirstCubistPass( 
                   cubist::params.handle, 
                   ray_origin, ray_direction,
                   0.01f,  // tmin       // TODO: smarter offset
                   1e16f,  // tmax
                   &payload );

    //
    // Update results
    // TODO: timview mode
    //
    const unsigned int image_index = launch_idx.y * launch_dims.x + launch_idx.x;
    float3             accum_color = payload.result;


    // CUBIST: the ULTIMATE cubist pass XD
    if( isSecondCubist && cubist::params.isCubistPassEnabled ) {

        float3   new_raydir = normalize (ray_direction + accum_color * 0.2);
    
        traceRadiance (
            cubist::params.handle,
            ray_origin,
            new_raydir,
            0.01f,  // tmin
            1e16f,  // tmax
            &payload );
        
        accum_color = payload.result;

    }

    if( subframe_index > 0 )
    {
        const float  a                = 1.0f / static_cast<float>( subframe_index + 1 );
        const float3 accum_color_prev = make_float3( cubist::params.accum_buffer[image_index] );
        accum_color                   = lerp( accum_color_prev, accum_color, a );
    }
    cubist::params.accum_buffer[image_index] = make_float4( accum_color, 1.0f );
    cubist::params.frame_buffer[image_index] = make_color( accum_color );
}

extern "C" __global__ void __miss__radiance() {
    
    // CUBIST: Not using env mapping. Return constant color
    if ( !cubist::params.isUseEnvMap ) 
    {
        cubist::setPayloadResult( cubist::params.miss_color );
        return;
    }

    const float3 ray_dir = optixGetWorldRayDirection();
    
    const float   theta  = atan2f( ray_dir.x, ray_dir.z );
    const float   phi    = M_PIf * 0.5f -  acosf( ray_dir.y );
    const float   u      = (theta + M_PIf) * (0.5f * M_1_PIf);
    const float   v      = 0.5f * ( 1.0f + sin(phi) );
    const float3  result = make_float3( tex2D<float4>(cubist::params.env_texture, u, v) );
    
    // CUBIST: enable second pass?
    if( cubist::params.isCubistPassEnabled )
        optixSetPayload_4 ( 1 );
    else 
        optixSetPayload_4 ( 0 );
    
    cubist::setPayloadResult( result );
}


extern "C" __global__ void __closesthit__occlusion()
{
    cubist::setPayloadOcclusion( true );
}


extern "C" __global__ void __closesthit__radiance()
{
    const cubist::HitGroupData* hit_group_data = reinterpret_cast<cubist::HitGroupData*>( optixGetSbtDataPointer() );
    const LocalGeometry          geom           = getLocalGeometry( hit_group_data->geometry_data );

    //
    // Retrieve material data
    //
    float3 base_color = make_float3( hit_group_data->material_data.pbr.base_color );
    if( hit_group_data->material_data.pbr.base_color_tex )
        base_color *= cubist::linearize(
            make_float3( tex2D<float4>( hit_group_data->material_data.pbr.base_color_tex, geom.UV.x, geom.UV.y ) ) );

    float  metallic  = hit_group_data->material_data.pbr.metallic;
    float  roughness = hit_group_data->material_data.pbr.roughness;
    float4 mr_tex    = make_float4( 1.0f );
    if( hit_group_data->material_data.pbr.metallic_roughness_tex )
        // MR tex is (occlusion, roughness, metallic )
        mr_tex = tex2D<float4>( hit_group_data->material_data.pbr.metallic_roughness_tex, geom.UV.x, geom.UV.y );
    roughness *= mr_tex.y;
    metallic *= mr_tex.z;


    //
    // Convert to material params
    //
    const float  F0         = 0.04f;
    const float3 diff_color = base_color * ( 1.0f - F0 ) * ( 1.0f - metallic );
    const float3 spec_color = lerp( make_float3( F0 ), base_color, metallic );
    const float  alpha      = roughness * roughness;

    // CUBIST: enable second pass?
    if( cubist::params.isCubistPassEnabled )
        optixSetPayload_4 ( 1 );

    // CUBIST: color edge by thredshold
    if( cubist::params.isEdgeEnabled ) {
        
        float3 result = make_float3( 0.0f );

        const float3 V       = -normalize( optixGetWorldRayDirection() );
        const float3 N       = geom.N;
        const float  N_dot_V = dot( N, V );
        
        // edge condition
        if ( N_dot_V < cubist::params.edge_threshold ) 
        {   
            // this will set u3, which will be assigned to params.isCubistPass
            optixSetPayload_4 ( 0 );
            
            // constant color on edges when debug mode
            if ( cubist::params.isDebugMode ) 
            {
                result = cubist::params.debug_color_a;
                cubist::setPayloadResult( result );
                return;   
            }    
        }
        else if ( cubist::params.isDebugMode )
        {
            // result = cubist::params.debug_color_b;
            // cubist::setPayloadResult( result );
            // return;   
        }
    }

    // CUBIST: no extra shading computation if it is unlit
    if ( cubist::params.isUnlit ) {
        cubist::setPayloadResult( make_float3( base_color.x, base_color.y, base_color.z ) );
        return;
    }

    //
    // compute direct lighting
    //

    float3 N = geom.N;
    if( hit_group_data->material_data.pbr.normal_tex )
    {
        const float4 NN =
            2.0f * tex2D<float4>( hit_group_data->material_data.pbr.normal_tex, geom.UV.x, geom.UV.y ) - make_float4( 1.0f );
        N = normalize( NN.x * normalize( geom.dpdu ) + NN.y * normalize( geom.dpdv ) + NN.z * geom.N );
    }

    float3 result = make_float3( 0.0f );

    for( int i = 0; i < cubist::params.lights.count; ++i )
    {
        Light light = cubist::params.lights[i];
        if( light.type == Light::Type::POINT )
        {
            // TODO: optimize
            const float  L_dist  = length( light.point.position - geom.P );
            const float3 L       = ( light.point.position - geom.P ) / L_dist;
            const float3 V       = -normalize( optixGetWorldRayDirection() );
            const float3 H       = normalize( L + V );
            const float  N_dot_L = dot( N, L );
            const float  N_dot_V = dot( N, V );
            const float  N_dot_H = dot( N, H );
            const float  V_dot_H = dot( V, H );

            if( N_dot_L > 0.0f && N_dot_V > 0.0f )
            {
                const float tmin     = 0.001f;           // TODO
                const float tmax     = L_dist - 0.001f;  // TODO
                const bool  occluded = cubist::traceOcclusion( cubist::params.handle, geom.P, L, tmin, tmax );
                if( !occluded )
                {
                    const float3 F     = cubist::schlick( spec_color, V_dot_H );
                    const float  G_vis = cubist::vis( N_dot_L, N_dot_V, alpha );
                    const float  D     = cubist::ggxNormal( N_dot_H, alpha );

                    const float3 diff = ( 1.0f - F ) * diff_color / M_PIf;
                    const float3 spec = F * G_vis * D;
                    
                    result += light.point.color * light.point.intensity * N_dot_L * ( diff + spec );
                }
            }
        }
        else if( light.type == Light::Type::AMBIENT )
        {
            result += light.ambient.color * base_color;
        }
    }

    cubist::setPayloadResult( result );
}
